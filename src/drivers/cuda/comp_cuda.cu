#include "hip/hip_runtime.h"
/*
 * StarPU
 * Copyright (C) INRIA 2008-2009 (see AUTHORS file)
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <stdint.h>
#include "comp_cuda.h"
//#include <core/jobs.h>
#include <common/parameters.h>

//#define MATA(x,y)	(datamatA[(x)+(y)*widthA])
//#define MATB(x,y)	(datamatB[(x)+(y)*widthB])
//#define MATC(x,y)	(datamatC[(x)+(y)*widthC])

#define MATA(x,y)	(datamatA[(x)+__mul24((y),widthA)])
#define MATB(x,y)	(datamatB[(x)+__mul24((y),widthB)])
#define MATC(x,y)	(datamatC[(x)+__mul24((y),widthC)])

//#define DEBUG
#define CHUNCKDEPTH	16 

#define MAXHEIGTH	(GRAIN/GRIDDIMY)
#define MAXWIDTH	(GRAIN/GRIDDIMX)

__shared__ float Achunk[CHUNCKDEPTH*MAXHEIGTH];
__shared__ float Bchunk[MAXWIDTH*CHUNCKDEPTH];
__shared__ float Cchunk[MAXWIDTH*MAXHEIGTH];

extern "C"
__global__ void 
cuda_mult
(
	float * datamatA, unsigned widthA, unsigned xaA,
	float * datamatB, unsigned widthB, unsigned yaB, unsigned ybB,
	float * datamatC, unsigned widthC, unsigned xaC, unsigned xbC, unsigned yaC, unsigned ybC
#ifdef DEBUG
	,int *toto
#endif
)
{	

	unsigned x,y;//,z;
	unsigned localx, localy, localz;
	unsigned nextz;

	int blockwidth = UPDIV( xbC - xaC , gridDim.x );
	int blockheigth = UPDIV( ybC - yaC , gridDim.y );

	int startx = MIN(xaC + blockIdx.x * blockwidth,  xbC);
	int endx   = MIN(xaC + (blockIdx.x+1) * blockwidth,  xbC);
	
	int starty = MIN(yaC + blockIdx.y * blockheigth, ybC);
	int endy   = MIN(yaC + (blockIdx.y+1) * blockheigth, ybC);


	int actual_width = (endx - startx);
	int actual_heigth = (endy - starty);

	/* zero the Cchunk ... */
	int i;
	for (i = threadIdx.x; i < actual_width*actual_heigth; i+= blockDim.x)
	{
		Cchunk[i] = 0;
	}

	__syncthreads();


	/* perform the actual computation */
	for (localz = 0 ; localz < ybB-yaB ; localz += CHUNCKDEPTH)
	{

		/* assert : ybB - yaB == xbA - xaA */
		nextz = MIN(localz+CHUNCKDEPTH, ybB-yaB);

		/* copy local A chunk */
		for (y = starty + threadIdx.y, localy = threadIdx.y;
		     y < endy ;
		     y += blockDim.y, localy += blockDim.y)
		{
			for (x = xaA + localz + threadIdx.x, localx = threadIdx.x; 
			     x < xaA + nextz;
			     x += blockDim.x, localx += blockDim.x)
			{
				//Achunk[localx + localy * CHUNCKDEPTH] = MATA(x, y);
				Achunk[localx + __mul24(localy, CHUNCKDEPTH)] = MATA(x, y);
			}
		}

		/* copy local B chunk */
		for (y = yaB + localz + threadIdx.y, localy = threadIdx.y;
		     y < yaB + nextz;
		     y += blockDim.y, localy += blockDim.y)
		{
			for (x = startx + threadIdx.x, localx = threadIdx.x;
			     x < endx ;
			     x += blockDim.x, localx += blockDim.x)
			{
				//Bchunk[localx + localy*MAXWIDTH] = MATB(x, y);
				Bchunk[localx + __mul24(localy, MAXWIDTH)] = MATB(x, y);
			}
		}

		__syncthreads();

		/* multiply both chunks */
		int index;
		for (localy = threadIdx.y; localy < actual_heigth ; localy += blockDim.y)
		{
			for (localx = threadIdx.x ; localx < actual_width ; localx += blockDim.x) 
			{
				for (index = 0; index < (nextz - localz) ; index++) 
				{
					Cchunk[localx + __umul24(localy, MAXWIDTH)] += 
								__mul24(Achunk[index + localy * CHUNCKDEPTH],
									Bchunk[localx + index * MAXWIDTH]);
				//	Cchunk[localx + localy * MAXWIDTH] +=  
				//		Achunk[index + localy * CHUNCKDEPTH] 
				//		* Bchunk[localx + index * MAXWIDTH];
				}	
			}
		}
		__syncthreads();
	}


	/* put Cchunk back into device memory */
	for (localy = threadIdx.y ; localy < actual_heigth ; localy += blockDim.y)
	{
		for (localx = threadIdx.x; localx < actual_width; localx += blockDim.x)
		{
			MATC(startx + localx, starty + localy) = Cchunk[localx + localy * MAXWIDTH];
		}
	}

	return;
}


