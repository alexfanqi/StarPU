#include "hip/hip_runtime.h"
/*
 * StarPU
 * Copyright (C) Université Bordeaux 1, CNRS 2008-2010 (see AUTHORS file)
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>

static __global__ void _cuda_unsigned_inc(unsigned *val)
{
	val[0]++;
}

extern "C" void cuda_codelet_unsigned_inc(void *descr[], STARPU_ATTRIBUTE_UNUSED void *cl_arg)
{
	unsigned *val = (unsigned *)STARPU_VARIABLE_GET_PTR(descr[0]);

	_cuda_unsigned_inc<<<1,1>>>(val);

	hipDeviceSynchronize();
}
