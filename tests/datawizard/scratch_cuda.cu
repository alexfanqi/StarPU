#include "hip/hip_runtime.h"
/*
 * StarPU
 * Copyright (C) INRIA 2008-2009 (see AUTHORS file)
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <stdio.h>
#include <starpu.h>

#define MAXNBLOCKS		32
#define MAXTHREADSPERBLOCK	128

static __global__ void increment_vector(unsigned *v, unsigned *tmp, int nx)
{
	const int tid = threadIdx.x + blockIdx.x*blockDim.x;
	const int nthreads = gridDim.x * blockDim.x;

	int i;
	for (i = tid; i < nx; i += nthreads)
	{
		v[i] = tmp[i] + 1;
	}
}

extern "C" void cuda_f(void *descr[], STARPU_ATTRIBUTE_UNUSED void *_args)
{
	unsigned *v = (unsigned *)STARPU_VECTOR_GET_PTR(descr[0]);
	unsigned *tmp = (unsigned *)STARPU_VECTOR_GET_PTR(descr[1]);

	unsigned nx = STARPU_VECTOR_GET_NX(descr[0]);
	size_t elemsize = STARPU_VECTOR_GET_ELEMSIZE(descr[0]);
	
	hipMemcpy(tmp, v, nx*elemsize, hipMemcpyDeviceToDevice);

	unsigned nblocks = 128;
	unsigned nthread_per_block = STARPU_MIN(MAXTHREADSPERBLOCK, (nx / nblocks));
	
	increment_vector<<<nblocks, nthread_per_block>>>(v, tmp, nx);
	hipDeviceSynchronize();
}
