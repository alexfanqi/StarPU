#include "hip/hip_runtime.h"
/*
 * StarPU
 * Copyright (C) INRIA 2008-2009 (see AUTHORS file)
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>

static __global__ void cuda_incrementer(unsigned *token)
{
	(*token)++;
}

extern "C" void increment_cuda(void *descr[], __attribute__ ((unused)) void *_args)
{
	unsigned *tokenptr = (unsigned *)STARPU_VECTOR_GET_PTR(descr[0]);

	cuda_incrementer<<<1,1>>>(tokenptr);
}
