#include "hip/hip_runtime.h"
/*
 * StarPU
 * Copyright (C) INRIA 2008-2009 (see AUTHORS file)
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>

static __global__ void cuda_variable(float * tab)
{
	*tab += 1.0;
	return;
}

extern "C" void cuda_codelet(void *descr[], STARPU_ATTRIBUTE_UNUSED void *_args)
{
	float *val = (float *)STARPU_VARIABLE_GET_PTR(descr[0]);

	cuda_variable<<<1,1>>>(val);
}
