#include "hip/hip_runtime.h"
/*
 * StarPU
 * Copyright (C) INRIA 2008-2009 (see AUTHORS file)
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>

static __global__ void cuda_incrementer(float * tab)
{
	tab[0] = tab[0] + 1.0;
	tab[2] = tab[2] + 1.0;
	
	return;
}

extern "C" void cuda_codelet(void *descr[], __attribute__ ((unused)) void *_args)
{
	float *val = (float *)STARPU_GET_VECTOR_PTR(descr[0]);

	cuda_incrementer<<<1,1>>>(val);
}
